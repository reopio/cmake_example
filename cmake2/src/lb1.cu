#include "hip/hip_runtime.h"
#include <iostream>

#include "lb1.h"

#include <hipblas.h>

__global__ void add_vec(float *a, float *b, float *c, int32_t n){

    int32_t i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<n){
        c[i] = a[i] + b[i];
    }

}


void test_cuda(){

    std::cout<<"this is a cuda test for cmake"<<std::endl;
    std::cout<<"this example calculate vec + vec using gpu"<<std::endl
    <<"with both kernel and cublas"<<std::endl;

    int32_t n=3;

    float *a,*b,*c,*ad,*bd,*cd;
    a=new float[n];
    b=new float[n];
    c=new float[n];

    a[0] = 0.1;
    a[1] = 0.2;
    a[2] = 0.3;
    b[0] = 0.4;
    b[1] = 0.5;
    b[2] = 0.6;

    hipMalloc((void**)&ad,sizeof(float)*n);
    hipMalloc((void**)&bd,sizeof(float)*n);
    hipMalloc((void**)&cd,sizeof(float)*n);

    hipMemcpy(ad,a,sizeof(float)*n,hipMemcpyHostToDevice);
    hipMemcpy(bd,b,sizeof(float)*n,hipMemcpyHostToDevice);

    add_vec<<<1,512>>>(ad,bd,cd,n);

    hipDeviceSynchronize();

    hipMemcpy(c,cd,sizeof(float)*n,hipMemcpyDeviceToHost);

    std::cout<<"vec a is [0.1, 0.2, 0.3]"<<std::endl;
    std::cout<<"vec b is [0.4, 0.5, 0.6]"<<std::endl;
    std::cout<<"a + b is [" <<c[0]<<", "
                            <<c[1]<<", "
                            <<c[2]
                            <<"]"<<std::endl;
    std::cout<<"calculated by cuda kernel"  <<std::endl
                                            <<std::endl
                                            <<std::endl;
    
    float alpha=1.0;

    hipblasSaxpy(n,
                alpha,
                ad,1,
                bd,1);

    hipMemcpy(b,bd,sizeof(float)*n,hipMemcpyDeviceToHost);
    std::cout<<"vec a is [0.1, 0.2, 0.3]"<<std::endl;
    std::cout<<"vec b is [0.4, 0.5, 0.6]"<<std::endl;
    std::cout<<"a + b is [" <<b[0]<<", "
                            <<b[1]<<", "
                            <<b[2]
                            <<"]"<<std::endl;
    std::cout<<"calculated by cublas"   <<std::endl;

    delete(a);
    delete(b);
    delete(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);

}